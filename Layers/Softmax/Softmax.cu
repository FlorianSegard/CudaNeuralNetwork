#include "hip/hip_runtime.h"
#include "Softmax.hpp"

// Forward Kernel for Softmax
__global__ void softmaxForwardKernel(float* input, float* output, int width, int height, size_t inStride, size_t outStride) {
    extern __shared__ float sharedData[];

    int row = blockIdx.x; // Each block handles one row
    int tid = threadIdx.x;

    if (row >= height) return;

    float* rowInput = input + row * inStride / sizeof(float);
    float* rowOutput = output + row * outStride / sizeof(float);

    // Step 1: Find the maximum value for numerical stability
    float maxVal = -3.402823466e+38F; // Smallest float value
    for (int i = tid; i < width; i += blockDim.x) {
        maxVal = fmaxf(maxVal, rowInput[i]);
    }

    // Perform reduction to find the maximum value across threads
    sharedData[tid] = maxVal;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] = fmaxf(sharedData[tid], sharedData[tid + s]);
        }
        __syncthreads();
    }
    maxVal = sharedData[0]; // Max value for the row

    // Step 2: Compute exponentials and their sum
    float sum = 0.0f;
    for (int i = tid; i < width; i += blockDim.x) {
        // printf("%f\n", rowInput[i]);
        rowOutput[i] = expf(rowInput[i] - maxVal); // Subtract maxVal for numerical stability
        sum += rowOutput[i];
    }

    // Perform reduction to calculate the sum of exponentials
    sharedData[tid] = sum;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }
    sum = sharedData[0]; // Total sum of exponentials

    // Step 3: Normalize each value
    for (int i = tid; i < width; i += blockDim.x) {
        rowOutput[i] /= sum;
    }
}

// Backward Kernel for Softmax
__global__ void softmaxBackwardKernel(float* output, float* dOutput, float* dInput, int width, int height,
                                     size_t outStride, size_t dOutStride, size_t dInStride) {
    int row = blockIdx.x;
    int tid = threadIdx.x;

    if (row >= height) return;

    extern __shared__ float sharedData[];
    float* rowOutput = output + (row * outStride / sizeof(float));
    float* rowDOutput = dOutput + (row * dOutStride / sizeof(float));
    float* rowDInput = dInput + (row * dInStride / sizeof(float));

    // 1. Compute sum of y_i * dL/dy_i for this row
    float sum = 0.0f;
    for (int i = tid; i < width; i += blockDim.x) {
        sum += rowOutput[i] * rowDOutput[i];
    }

    // Parallel reduction to get the total sum
    sharedData[tid] = sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }
    sum = sharedData[0];

    // 2. Compute final gradients
    // dL/dx_i = y_i * (dL/dy_i - sum)
    for (int i = tid; i < width; i += blockDim.x) {
        float yi = rowOutput[i];
        rowDInput[i] = yi * (rowDOutput[i] - sum);
    }
}

// Forward Pass on GPU
Tensor<float> softmaxGPU(Tensor<float>& input) {
    Tensor<float> output(input.width, input.height, true);

    int blockSize = 256;
    int gridSize = input.height;
    size_t sharedMemSize = blockSize * sizeof(float);

    softmaxForwardKernel<<<gridSize, blockSize, sharedMemSize>>>(
        input.buffer, output.buffer, input.width, input.height, input.stride, output.stride);

    hipDeviceSynchronize();
    return output;
}

// Backward Pass on GPU
Tensor<float> softmaxBackwardGPU(Tensor<float>& output, Tensor<float>& dOutput) {
    Tensor<float> dInput(output.width, output.height, true);

    int blockSize = 256;
    int gridSize = output.height;

    softmaxBackwardKernel<<<gridSize, blockSize>>>(
        output.buffer, dOutput.buffer, dInput.buffer, output.width, output.height, output.stride, dOutput.stride, dInput.stride);

    hipDeviceSynchronize();
    return dInput;
}
