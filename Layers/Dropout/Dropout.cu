#include "hip/hip_runtime.h"
// dropout.cu

#include "Dropout.hpp"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <ctime>


hiprandState* d_states = nullptr;




__global__ void initCurandStates(hiprandState* states, unsigned long seed, int width, int height, size_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * (stride / sizeof(float)) + x;
        hiprand_init(seed, index, 0, &states[index]);
    }
}

void initializeCurandStates(int width, int height, size_t stride) {
    if (!d_states) {
        hipMalloc(&d_states, width * height * sizeof(hiprandState));

        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                      (height + blockSize.y - 1) / blockSize.y);

        unsigned long seed = static_cast<unsigned long>(time(NULL));
        initCurandStates<<<gridSize, blockSize>>>(d_states, seed, width, height, stride);
        hipDeviceSynchronize();
    }
}

void freeCurandStates() {
    if (d_states) {
        hipFree(d_states);
        d_states = nullptr;
    }
}

__global__ void fillMaskKernel(float* mask, hiprandState* states, float drop_rate, int width, int height, size_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * (stride / sizeof(float)) + x;
        float rand_val = hiprand_uniform(&states[index]);
        mask[index] = (rand_val > drop_rate) ? 1.0f : 0.0f;
    }
}

void fillMaskGPU(Tensor<float>* mask, float drop_rate) {
    int width = mask->width;
    int height = mask->height;
    size_t stride = mask->stride;


    // Define grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Initialize hiprand states
    initializeCurandStates(width, height, stride);

    // Fill the mask with random values
    fillMaskKernel<<<gridSize, blockSize>>>(mask->buffer, d_states, drop_rate, width, height, stride);
    hipDeviceSynchronize();

}



