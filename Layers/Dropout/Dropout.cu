#include "hip/hip_runtime.h"
// dropout.cu

#include "Dropout.hpp"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <ctime>


__global__ void fillMaskKernel(float* mask, hiprandState* states, float drop_rate,
                              int width, int height, size_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * (stride / sizeof(float)) + x;
        float rand_val = hiprand_uniform(&states[index]);
        mask[index] = (rand_val >= drop_rate) ? 1.0f : 0.0f;
    }
}

void fillMaskGPU(Tensor<float>* mask, float drop_rate, hiprandState* states) {
    dim3 blockSize(32, 32);
    dim3 gridSize((mask->width + blockSize.x - 1) / blockSize.x,
                  (mask->height + blockSize.y - 1) / blockSize.y);

    fillMaskKernel<<<gridSize, blockSize>>>(mask->buffer, states, drop_rate,
                                           mask->width, mask->height, mask->stride);
    hipDeviceSynchronize();
}

__global__ void initCurandStates(hiprandState* states, unsigned long seed, int width, int height, size_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * (stride / sizeof(float)) + x;
        hiprand_init(seed + index, 0, 0, &states[index]);
    }
}

void initializeCurandStates(hiprandState** d_states, int width, int height, size_t stride) {
    size_t total_elements = (stride / sizeof(float)) * height;
    hipMalloc(d_states, total_elements * sizeof(hiprandState));

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    auto seed = static_cast<unsigned long>(time(nullptr));
    initCurandStates<<<gridSize, blockSize>>>(*d_states, seed, width, height, stride);
    hipDeviceSynchronize();
}