#include "hip/hip_runtime.h"
#include "Tensor.hpp"

// ----------------------------------------------------------- TRANSPOSE ----------------------------------------------------------- \\

// Very simple transpose kernel might not be optimized
template <class T>
__global__ void transposeKernel(const T* input, T* output, int width, int height, size_t inStride, size_t outStride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        output[x * outStride / sizeof(T) + y] = input[y * inStride / sizeof(T) + x];
    }
}

template <class T>
Tensor<T> transposeGPU(const Tensor<T>& input) {
    Tensor<T> result(input.height, input.width, true);

    dim3 blockSize(32, 32);
    dim3 gridSize((input.width + blockSize.x - 1) / blockSize.x,
                  (input.height + blockSize.y - 1) / blockSize.y);

    transposeKernel<T><<<gridSize, blockSize>>>(
        input.buffer, result.buffer, input.width, input.height, input.stride, result.stride
    );
    hipDeviceSynchronize();

    return result;
}

// template definitions
template Tensor<float> transposeGPU(const Tensor<float>& input);
template Tensor<double> transposeGPU(const Tensor<double>& input);
template Tensor<int> transposeGPU(const Tensor<int>& input);

// ----------------------------------------------------------- FILL UP WITH ZEROS ----------------------------------------------------------- \\

// Very simple filling up with zeros kernel might not be optimized
template <class T>
__global__ void fillZeroKernel(T* input, int width, int height, size_t inStride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        input[x + inStride / sizeof(T) * y] = 0;
    }
}

template <class T>
void fillZeroGPU(Tensor<T>& input) {

    dim3 blockSize(32, 32);
    dim3 gridSize((input.width + blockSize.x - 1) / blockSize.x,
                  (input.height + blockSize.y - 1) / blockSize.y);

    fillZeroKernel<T><<<gridSize, blockSize>>>(
        input.buffer, input.width, input.height, input.stride
    );
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

}

// template definitions
template void fillZeroGPU(Tensor<float>& input);
template void fillZeroGPU(Tensor<double>& input);
template void fillZeroGPU(Tensor<int>& input);


// ----------------------------------------------------------- DOT ----------------------------------------------------------- \\


// Very simple filling up with zeros kernel might not be optimized
// template <class T>
// __global__ void dotGPUKernel(T* input, T* other, T* result, int width_input, int height_input, int width_output, size_t inputStride, size_t otherStride, size_t resultStride) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (y < height_input && x < width_output) {
//         T sum = 0;
//         for (int k = 0; k < width_input; k++)
//         {
//             T a_val = input[k + y * inputStride / sizeof(T)];
//             T b_val = other[x + k * otherStride / sizeof(T)];
//             sum += a_val * b_val;
//         }
//         result[x + y * otherStride / sizeof(T)] = sum;
//     }
// }


#define TILE_SIZE 16

template <class T>
__global__ void dotGPUKernel(T* input, T* other, T* result,
                                      int width_input, int height_input, int width_output,
                                      size_t inputStride, size_t otherStride, size_t resultStride) {
    __shared__ T shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ T shared_B[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    T sum = 0;

    size_t inputStrideElements = inputStride / sizeof(T);
    size_t otherStrideElements = otherStride / sizeof(T);
    size_t resultStrideElements = resultStride / sizeof(T);

    for (int t = 0; t < (width_input + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load tile from input matrix into shared memory
        if (row < height_input && t * TILE_SIZE + tx < width_input) {
            shared_A[ty][tx] = input[row * inputStrideElements + t * TILE_SIZE + tx];
        } else {
            shared_A[ty][tx] = 0;
        }

        // Load tile from other matrix into shared memory
        if (col < width_output && t * TILE_SIZE + ty < width_input) {
            shared_B[ty][tx] = other[(t * TILE_SIZE + ty) * otherStrideElements  + col];
        } else {
            shared_B[ty][tx] = 0;
        }

        __syncthreads();

        // Compute partial product for the tile
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += shared_A[ty][k] * shared_B[k][tx];
        }

        __syncthreads();
    }

    if (row < height_input && col < width_output) {
        result[row * resultStrideElements + col] = sum;
    }
}


template <class T>
Tensor<T> dotGPU(const Tensor<T>& input, const Tensor<T>& other) {
    dim3 blockSize(16, 16);
    dim3 gridSize((input.width + blockSize.x - 1) / blockSize.x,
                  (input.height + blockSize.y - 1) / blockSize.y);

    Tensor<T> result(other.width, input.height, true);

    dotGPUKernel<T><<<gridSize, blockSize>>>(
        input.buffer, other.buffer, result.buffer, 
        input.width, input.height, other.width, 
        input.stride, other.stride, result.stride
    );
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return result;
}

// template definitions
template Tensor<float> dotGPU(const Tensor<float>& input, const Tensor<float>& other);
template Tensor<double> dotGPU(const Tensor<double>& input, const Tensor<double>& other);
template Tensor<int> dotGPU(const Tensor<int>& input, const Tensor<int>& other);